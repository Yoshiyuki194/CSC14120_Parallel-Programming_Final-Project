#include "hip/hip_runtime.h"
#include "../../kernel_launcher.h"
#include "device.h"

#define TILE_WIDTH 32

// Convolution forward kernel: Shared memory implementation
__global__ void smem_conv_forward_kernel(const float *in, float *out, const float *weight,
                                      const int channel_in, const int channel_out,
                                      const int height_in, const int width_in, const int kernel_width)
{
    const int height_out = height_in - kernel_width + 1;
    const int width_out = width_in - kernel_width + 1;

    int height_grid = (height_out - 1) / TILE_WIDTH + 1;
    int width_grid = (width_out - 1) / TILE_WIDTH + 1;

    int sample_idx = blockIdx.x;
    int map_idx = blockIdx.y;
    int row_base = (blockIdx.z / width_grid) * TILE_WIDTH;
    int col_base = (blockIdx.z % width_grid) * TILE_WIDTH;
    int row = row_base + threadIdx.y;
    int col = col_base + threadIdx.x;

    extern __shared__ float shmem[];
    float* in_shared = &shmem[0];
    float* weight_shared = &shmem[(TILE_WIDTH + kernel_width - 1) * (TILE_WIDTH + kernel_width - 1)];

    float accum = 0;

    if (row >= height_out || col >= width_out)
        return;

    int hw_in = height_in * width_in;
    int hw_out = height_out * width_out;

    for (int c = 0; c < channel_in; c++)
    {
        // Load weight into shared memory
        if (threadIdx.y < kernel_width && threadIdx.x < kernel_width)
            weight_shared[threadIdx.y * kernel_width + threadIdx.x] = weight[map_idx * channel_in * kernel_width * kernel_width + c * kernel_width * kernel_width + threadIdx.y * kernel_width + threadIdx.x];
        __syncthreads();
        // Load input into shared memory
        for (int i = row; i < row_base + kernel_width - 1; i += TILE_WIDTH)
            for (int j = col; j < col_base + kernel_width - 1; j += TILE_WIDTH)
                if (i < height_in && j < width_in)
                    in_shared[(i - row_base + threadIdx.y) * (TILE_WIDTH + kernel_width - 1) + (j - col_base + threadIdx.x)] = in[sample_idx * channel_in * hw_in + c * hw_in + i * width_in + j];
        __syncthreads();
        // Compute convolution
        for (int p = 0; p < kernel_width; p++)
            for (int q = 0; q < kernel_width; q++)
                if (row + p < height_out && col + q < width_out)
                    accum += in_shared[threadIdx.y * (TILE_WIDTH + kernel_width - 1) + threadIdx.x + p * (TILE_WIDTH + kernel_width - 1) + q] * weight_shared[p * kernel_width + q];
        __syncthreads();
    }
    // Store output
    out[sample_idx * channel_out * hw_out + map_idx * hw_out + row * width_out + col] = accum;
}

void KernelLauncher::smem_forward(const float *in, float *out, const float *weight,
                                         const int n_samples, const int channel_in, const int channel_out,
                                         const int height_in, const int width_in, const int kernel_width, const int n_streams)
{
    int height_out = height_in - kernel_width + 1;
    int width_out = width_in - kernel_width + 1;
    int size_in = n_samples * channel_in * height_in * width_in;
    int size_out = n_samples * channel_out * height_out * width_out;
    int size_weight = channel_out * channel_in * kernel_width * kernel_width;

    float *d_in;
    float *d_out;
    float *d_weight;
    CHECK(hipMalloc((void **)&d_in, size_in * sizeof(float)));
    CHECK(hipMalloc((void **)&d_out, size_out * sizeof(float)));
    CHECK(hipMalloc((void **)&d_weight, size_weight * sizeof(float)));
    CHECK(hipMemcpy(d_in, in, size_in * sizeof(float), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_weight, weight, size_weight * sizeof(float), hipMemcpyHostToDevice));

    // Create "nStreams" device streams
    hipStream_t *streams = (hipStream_t *)malloc(n_streams * sizeof(hipStream_t));
    for (int i = 0; i < n_streams; i++)
        CHECK(hipStreamCreate(&(streams[i])));
    
    int n_samples_per_stream = (n_samples + n_streams - 1) / n_streams;
    int size_in_per_stream = n_samples_per_stream * channel_in * height_in * width_in;
    int size_out_per_stream = n_samples_per_stream * channel_out * height_out * width_out;
    int size_weight_per_stream = channel_out * channel_in * kernel_width * kernel_width;
    int size_in_per_sample = channel_in * height_in * width_in;
    int size_out_per_sample = channel_out * height_out * width_out;
    int size_weight_per_sample = channel_out * channel_in * kernel_width * kernel_width;

    // Set grid and block dimensions and launch the kernel
    int height_grid = (height_out - 1) / TILE_WIDTH + 1;
    int width_grid = (width_out - 1) / TILE_WIDTH + 1;
    int z = height_grid * width_grid;

    for (int i = 0; i < n_streams; i++)
    {
        int offset = i * n_samples_per_stream;
        int n_samples_per_stream_ = min(n_samples - offset, n_samples_per_stream);
        int size_in_per_stream = n_samples_per_stream_ * channel_in * height_in * width_in;
        int size_out_per_stream = n_samples_per_stream_ * channel_out * height_out * width_out;
        int size_in_per_sample = channel_in * height_in * width_in;
        int size_out_per_sample = channel_out * height_out * width_out;

        CHECK(hipMemcpyAsync(d_in + offset * size_in_per_sample, in + offset * size_in_per_sample, size_in_per_stream * sizeof(float), hipMemcpyHostToDevice, streams[i]));
        CHECK(hipMemcpyAsync(d_weight, weight, size_weight * sizeof(float), hipMemcpyHostToDevice, streams[i]));

        dim3 dimGrid(n_samples_per_stream_, channel_out, z);
        dim3 dimBlock(TILE_WIDTH, TILE_WIDTH, 1);
        size_t shmem_size = sizeof(float) * ((TILE_WIDTH + kernel_width - 1) * (TILE_WIDTH + kernel_width - 1) + kernel_width * kernel_width);
        smem_conv_forward_kernel<<<dimGrid, dimBlock, shmem_size, streams[i]>>>(d_in + offset * size_in_per_sample, d_out + offset * size_out_per_sample, d_weight, channel_in, channel_out, height_in, width_in, kernel_width);
        CHECK(hipMemcpyAsync(out + offset * size_out_per_sample, d_out + offset * size_out_per_sample, size_out_per_stream * sizeof(float), hipMemcpyDeviceToHost, streams[i]));
    }

    // Destroy device streams
    for (int i = 0; i < n_streams; i++)
    {
        CHECK(hipStreamSynchronize(streams[i]));
        CHECK(hipStreamDestroy(streams[i]));
    }

    CHECK(hipDeviceSynchronize());
    CHECK(hipGetLastError());
    
    CHECK(hipMemcpy(out, d_out, size_out * sizeof(float), hipMemcpyDeviceToHost));
    CHECK(hipFree(d_in));
    CHECK(hipFree(d_out));
    CHECK(hipFree(d_weight));
    free(streams);
}